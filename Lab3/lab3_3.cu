#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include "kmeans.h"

#define MAX_BLOCK_SZ 128

template <int blockSize>
__device__ void warpReduce(volatile int* sdata,unsigned int tid) {
    if (blockSize >=  64) sdata[tid] += sdata[tid + 32];
    if (blockSize >=  32) sdata[tid] += sdata[tid + 16];
    if (blockSize >=  16) sdata[tid] += sdata[tid + 8];
    if (blockSize >=  8) sdata[tid] += sdata[tid + 4];
    if (blockSize >=  4) sdata[tid] += sdata[tid + 2];
    if (blockSize >=  2) sdata[tid] += sdata[tid + 1];
}

template <int blockSize>
__global__ void sum(int* g_odata, int* g_idata, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    int i = blockIdx.x*(blockSize*2) + tid;
    int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;
    while (i < n){
        sdata[tid] += g_idata[i];
        g_idata[i] = 0;
        if (i + blockSize < n) {
            sdata[tid] += g_idata[i+blockSize];  
            g_idata[i+blockSize] = 0;
        }
        i += gridSize;  
    }
    __syncthreads();
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
    if (tid == 0){
        g_odata[blockIdx.x] = sdata[0];
    } 
}

__device__ inline static float euclid_distance(int    numCoords,
                    int    numObjs,
                    int    numClusters,
                    int    tid,
                    int    clusterId,
                    float *objects,
                    float *clusters
                )
{
    float ans=0.0;
    for (int i = 0; i < numCoords; i++) {
        ans += (objects[3*tid+i] - clusters[i + clusterId*3]) *
               (objects[3*tid+i] - clusters[i + clusterId*3]);
    }

    return(ans);
}

__global__ static void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          float *objects,    
                          float *deviceClusters,
                          int *membership,
                          int *changedmembership
)
{
    extern __shared__ float sharedMem[];
    float *sh_Clusters = sharedMem;
    float *sh_Objects = (float*)&sh_Clusters[numClusters * 3];

    for(int i = 0; i < numCoords * numClusters; i++) {
        sh_Clusters[i] = deviceClusters[i];
    }
    __syncthreads();

    unsigned int tid = threadIdx.x;
    int objectId = blockDim.x * blockIdx.x + threadIdx.x;

    while (objectId < numObjs) {
        int   index, i;
        float dist, min_dist;
        
        for(int i = 0; i < numCoords; i++) { 
            sh_Objects[3*tid+i] = objects[3*objectId+i];
        }

        index = 0;
        min_dist = euclid_distance(numCoords, numObjs, numClusters, tid,
             0, sh_Objects, sh_Clusters);

        for (i=1; i<numClusters; i++) {
            dist = euclid_distance(numCoords, numObjs, numClusters, tid,
                i, sh_Objects, sh_Clusters);
            if (dist < min_dist) {
                min_dist = dist;
                index    = i;
            }
        }
        if (membership[objectId] != index)
        {
            changedmembership[objectId] = 1;
            membership[objectId] = index;
            
        }
        objectId += blockDim.x * gridDim.x;
    }
}

float** cuda_kmeans(float **objects,      /* in: [numObjs][numCoords] */
    int     numCoords,    
    int     numObjs,     
    int     numClusters, 
    int    *membership  
)
{
#pragma region declaration

    int      i, j, index, loop=0;
    int total_sum = 0;

    float error = 0.001;
    float delta;              /* % of objects change their clusters */
    int GRID_SZ = (numObjs+MAX_BLOCK_SZ-1)/ MAX_BLOCK_SZ;
    int *newClusterSize; /* objects assigned in each new cluster */
    float  **loopClusters;   /* [numClusters][numCoords] */
    float  **newClusters;  /* [numClusters][numCoords] */  

    /*DEVICE*/
    int* d_block_sums;
    int* d_total_sum;

    int *d_Membership;
    int *d_Changedmembership;
    float *d_Objects;
    float *d_Clusters;

#pragma endregion

#pragma region init

    gpuErrchk(hipSetDevice(0));
    
    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;

    /* pick first numClusters elements of objects[] as initial cluster centers*/
    malloc2D(loopClusters, numClusters ,numCoords , float);
    for (i = 0; i < numClusters; i++) {
        for (j = 0; j < numCoords; j++) {
            loopClusters[i][j] = objects[i][j];
        }
    }

    newClusterSize = (int*) malloc(numClusters* sizeof(int));
    assert(newClusterSize != NULL);

    malloc2D(newClusters, numClusters,numCoords, float);
    memset(newClusters[0], 0, (numCoords * numClusters) * sizeof(float));
    memset(newClusterSize, 0, numClusters * sizeof(int));

    gpuErrchk(hipMalloc(&d_Objects, numObjs*numCoords*sizeof(float)));
    gpuErrchk(hipMalloc(&d_Clusters, numClusters*numCoords*sizeof(float)));
    gpuErrchk(hipMalloc(&d_Membership, numObjs*sizeof(int)));
    gpuErrchk(hipMalloc(&d_Changedmembership, numObjs*sizeof(int)));
    gpuErrchk(hipMalloc(&d_block_sums, sizeof(int) * GRID_SZ));
    gpuErrchk(hipMalloc(&d_total_sum, sizeof(int)));

    gpuErrchk(hipMemset(d_total_sum, 0, sizeof(int)));
    gpuErrchk(hipMemset(d_block_sums, 0, sizeof(int) * GRID_SZ));
    gpuErrchk(hipMemset(d_Changedmembership,0, numObjs*sizeof(int)));
    
    gpuErrchk(hipMemcpy(d_Objects, objects[0], numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_Membership, membership, numObjs*sizeof(int), hipMemcpyHostToDevice));

#pragma endregion

do {

    int tot_cor = 0;
    gpuErrchk(hipMemcpy(d_Clusters, loopClusters[0], numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice));

    find_nearest_cluster
        <<<GRID_SZ, MAX_BLOCK_SZ,sizeof(float) * numCoords * numClusters + sizeof(float) * numCoords * MAX_BLOCK_SZ>>> //assume here that 1 object per 1 thread in a block
        (numCoords, numObjs, numClusters, d_Objects,d_Clusters, d_Membership, d_Changedmembership);

    sum<MAX_BLOCK_SZ><<<GRID_SZ, MAX_BLOCK_SZ, sizeof(int) * MAX_BLOCK_SZ>>>(d_block_sums, d_Changedmembership, numObjs);

    sum<MAX_BLOCK_SZ><<<1, MAX_BLOCK_SZ, sizeof(int) * MAX_BLOCK_SZ>>>(d_total_sum, d_block_sums, GRID_SZ);

    gpuErrchk(hipMemcpy(&total_sum, d_total_sum, sizeof(int), hipMemcpyDeviceToHost));

    delta = (float)total_sum/(float)numObjs;

    gpuErrchk(hipMemcpy(membership, d_Membership, numObjs*sizeof(int), hipMemcpyDeviceToHost));
   
    for (i=0; i<numObjs; i++) {
        /* find the array index of nestest cluster center */
        index = membership[i];
      
        /* update new cluster centers : sum of objects located within */
        newClusterSize[index] += 1;
        for (j=0; j<numCoords; j++)
            {
                newClusters[index][j] += objects[i][j];
            }
    }
    
    /*set new cluster centers*/
    for (i=0; i<numClusters; i++) {
        for (j=0; j<numCoords; j++) {
            if (newClusterSize[i] > 0)
            {             
                loopClusters[i][j] = newClusters[i][j] / newClusterSize[i];
            }
            newClusters[i][j] = 0.0;   /* set back to 0 */
        }
        tot_cor += newClusterSize[i];
        newClusterSize[i] = 0;   /* set back to 0 */
    }

    if (tot_cor != numObjs) {
        printf("Sum error \n");
        exit(-1);
    }

    } while (delta > error && loop++ < 500);

#pragma region free
    gpuErrchk(hipFree(d_Membership));
    gpuErrchk(hipFree(d_Changedmembership));
    gpuErrchk(hipFree(d_Objects));
    gpuErrchk(hipFree(d_Clusters));
    gpuErrchk(hipFree(d_total_sum));
	gpuErrchk(hipFree(d_block_sums));

    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);
#pragma endregion

    return loopClusters;
}